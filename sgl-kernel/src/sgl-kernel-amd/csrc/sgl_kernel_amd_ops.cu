#include "utils.hpp"

// moe_align_block_size
void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts, int64_t block_size,
                          torch::Tensor sorted_token_ids, torch::Tensor experts_ids, torch::Tensor num_tokens_post_pad,
                          torch::Tensor token_cnts_buffer, torch::Tensor cumsum_buffer);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // moe_align_block_size
  m.def("moe_align_block_size", &moe_align_block_size, "MOE Align Block Size (CUDA)");
}