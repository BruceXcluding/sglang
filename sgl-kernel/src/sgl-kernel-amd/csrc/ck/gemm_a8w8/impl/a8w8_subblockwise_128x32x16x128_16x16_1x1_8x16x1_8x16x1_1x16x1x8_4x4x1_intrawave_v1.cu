#include "../gemm_a8w8_subblock_common.cuh"

template <typename DEDataType, typename ABDataType>
torch::Tensor
a8w8_subblockwise_128x32x16x128_16x16_1x1_8x16x1_8x16x1_1x16x1x8_4x4x1_intrawave_v1(
    torch::Tensor& XQ,
    torch::Tensor& WQ,
    torch::Tensor& x_scale,
    torch::Tensor& w_scale,
    torch::Tensor& Y) 
{
  int M = size_to_dim_(XQ.dim() - 1, XQ.sizes());
  int N = WQ.size(0);
  int K = WQ.size(1);
  bool k_pad = (K % 128 != 0);
  bool m_pad = (M % 128 != 0);
  bool n_pad = (N % 128 != 0);
  using DeviceGemmInstance = DeviceGemmHelper<
    DEDataType, ABDataType,
    128,
    32,
    16,
    128,
    16,
    16,
    1,
    1,
    S<8, 16, 1>,
    S<8, 16, 1>,
    S<1, 16, 1, 8>,
    S<4, 4, 1>,
    ck::BlockGemmPipelineScheduler::Intrawave,
    ck::BlockGemmPipelineVersion::v1> ;
  return a8w8_subblockwise_128x16x32x128_16x16_1x1_8x16x1_8x16x1_1x16x1x8_4x4x1_intrawave_v1<DeviceGemmInstance>(XQ, WQ, x_scale, w_scale, Y);
}

template torch::Tensor
a8w8_subblockwise_128x32x16x128_16x16_1x1_8x16x1_8x16x1_1x16x1x8_4x4x1_intrawave_v1<B16, F8>(
    torch::Tensor& XQ,
    torch::Tensor& WQ,
    torch::Tensor& x_scale,
    torch::Tensor& w_scale,
    torch::Tensor& Y
)